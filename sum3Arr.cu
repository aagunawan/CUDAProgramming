#include "hip/hip_runtime.h"
%%cu
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <vector>
#include <iostream>
#include <cstdlib>
#include <functional>

using std::cout;
using std::vector;

__global__ void sum3Arr(const int* arr1, const int* arr2, const int* arr3, int* arr4){
    int blockOffset = blockIdx.x*blockDim.x;
    int gid = blockOffset + threadIdx.x;

    arr4[gid] = arr1[gid] + arr2[gid] + arr3[gid];
}

int main(){
    int N = 1 << 11;
    hipError_t error;

    vector<int> h_A(N*N);
    vector<int> h_B(N*N);
    vector<int> h_C(N*N);
    vector<int> h_D(N*N);
    vector<int> h_(N*N);

    for (int i = 0; i < N*N; i++){
        h_A[i] = i;
        h_B[i] = i;
        h_C[i] = i;
    }

    clock_t cpu_start, cpu_end;
    cpu_start = clock();
    for (int i = 0; i < N*N; i++){
        h_[i] = h_A[i] + h_B[i] + h_C[i];
    }
    cpu_end = clock();
    printf("Sum array CPU: %4.6f\n", (double)((double)(cpu_end - cpu_start)/CLOCKS_PER_SEC));

    int *d_a, *d_b, *d_c, *d_d;
    size_t byte_size = N*N*sizeof(int);
    error = hipMalloc((int**)&d_a, byte_size);
    if (error != hipSuccess){
        fprintf(stderr,"Error: %s \n", hipGetErrorString(error));
    }

    error = hipMalloc((int**)&d_b, byte_size);
    if (error != hipSuccess){
        fprintf(stderr,"Error: %s \n", hipGetErrorString(error));
    }
    error = hipMalloc((int**)&d_c, byte_size);
    if (error != hipSuccess){
        fprintf(stderr,"Error: %s \n", hipGetErrorString(error));
    }
    error = hipMalloc((int**)&d_d, byte_size);
    if (error != hipSuccess){
        fprintf(stderr,"Error: %s \n", hipGetErrorString(error));
    }

    clock_t htod_start, htod_end;
    htod_start = clock();
    hipMemcpy(d_a, h_A.data(), byte_size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_B.data(), byte_size, hipMemcpyHostToDevice);
    hipMemcpy(d_c, h_C.data(), byte_size, hipMemcpyHostToDevice);
    htod_end = clock();
    
    dim3 block(64);
    dim3 grid((N*N)/block.x);

    clock_t gpu_start, gpu_end;
    gpu_start = clock();
    sum3Arr<<<grid,block>>>(d_a, d_b, d_c, d_d);
    hipDeviceSynchronize();
    gpu_end = clock();

    clock_t dtoh_start, dtoh_end;
    dtoh_start = clock();
    hipMemcpy(h_D.data(), d_d, byte_size, hipMemcpyDeviceToHost);
    dtoh_end = clock();
    // for (int i = 0; i < 128; i++){
    //     cout << h_D[i] << std::endl;
    // }

    printf("htod: %4.6f\n", (double)((double)(htod_end - htod_start)/CLOCKS_PER_SEC));
    printf("gpu: %4.6f\n", (double)((double)(gpu_end - gpu_start)/CLOCKS_PER_SEC));
    printf("dtoh: %4.6f\n", (double)((double)(dtoh_end - dtoh_start)/CLOCKS_PER_SEC));
    

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    hipFree(d_d);
    hipDeviceReset();
    return 0;
}